/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


// System includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>

#define NUM_VERTICES 1024

#define SEQUENTIAL 0

#define POS(i, j) (((j) * NUM_VERTICES) + i)

#define PRINT_EDGES 0

#define PSEUDO_RANDOM_SIZE 1024

#define BLOCK_SIZE 32

#define DEBUG 1

// CUDA API error checking macro
static void handleError(hipError_t err,
                        const char *file,
                        int line ) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line );
        exit(EXIT_FAILURE);
    }
}
#define cudaCheck( err ) (handleError(err, __FILE__, __LINE__ ))

int sampleBinomialDistribution(double succesProbability) {
    return (int)((double)rand() / (double)RAND_MAX < succesProbability);
}

void printArray(int arr[]) {
    printf("Array(");
    for (int i = 0; i < NUM_VERTICES; i++) {
        if (arr[i] > 0) {
            printf("%d:%d, ", i, arr[i]);
        }
    }
    printf(")\n");
}

void printEdges(int *adjacencyMatrix) {
    printf("Edges: \n");
    for (int i = 0; i < NUM_VERTICES; i++) {
        for (int j = 0; j < NUM_VERTICES; j++) {
            if (adjacencyMatrix[POS(i, j)] == 1) {
                printf("\t(%d, %d)\n", i, j);
            }
        }
    }
}

void star(int *adjacencyMatrix) {
    memset(adjacencyMatrix, 0, NUM_VERTICES * NUM_VERTICES);
    for (int j = 1; j < NUM_VERTICES; j++) {
        adjacencyMatrix[POS(0, j)] = 1;
        adjacencyMatrix[POS(j, 0)] = 1;
    }
}

void cycle(int *adjacencyMatrix) {
    memset(adjacencyMatrix, 0, NUM_VERTICES * NUM_VERTICES);
    for (int i = 0; i < NUM_VERTICES; i++) {
        adjacencyMatrix[POS(i, (i + 1) % NUM_VERTICES)] = 1;
        adjacencyMatrix[POS(i, (i + NUM_VERTICES - 1) % NUM_VERTICES)] = 1;
    }
}

// Skewed random graph
void randomGraph(int *adjacencyMatrix)
{
    memset(adjacencyMatrix, 0, NUM_VERTICES * NUM_VERTICES);
    int edge;
    for (int i = 0; i < NUM_VERTICES; i++) {
        for (int j = i + 1; j < NUM_VERTICES; j++) {
            double probability = (((double) i) + j) / (4 * NUM_VERTICES);
            edge = sampleBinomialDistribution(probability);
            adjacencyMatrix[POS(i, j)] = edge;
            adjacencyMatrix[POS(j, i)] = edge;
        }
    }
}

void fillPseudoRandoms(float pseudoRandoms[]) {
    for (int i = 0; i < PSEUDO_RANDOM_SIZE; i++) {
        pseudoRandoms[i] = 1. * (i % 3) / 3.;
    }
}

bool verifyMaximalIndependentSet(int *adjacencyMatrix, int* independentSet) {
    for (int i = 0; i < NUM_VERTICES; i++) {
        int neighborsInV = 0;
        for (int j = 0; j < NUM_VERTICES; j++) {
            if (i != j && adjacencyMatrix[POS(i, j)] == 1 && independentSet[j] == 1) {
                if (independentSet[i]) {
                    printf("Set is not independent as it contains neighbors %d and %d", i, j);
                    return false;
                }
              neighborsInV++;
            }
        }
        if (independentSet[i] == 0 && neighborsInV == 0) {
            printf("Set is not maximal as %d (not in set) has no neighbors in set\n", i);
            return false;
        }
    }
    return true;
}

void lfIndependentSet(int *adjacencyMatrix, int* independentSet) {
    memset(independentSet, 0, NUM_VERTICES);

    for (int i = 0; i < NUM_VERTICES; i++) {
        bool hasNeighborsInV = false;
        for (int j = 0; j < i; j++) {
            if (adjacencyMatrix[POS(i, j)] == 1 && independentSet[j] == 1) {
                hasNeighborsInV = true;
            }
        }
        if (!hasNeighborsInV) {
            independentSet[i] = 1;
        }
    }
}

int main()
{
    srand(1);

    // Allocate host memory for matrices A, B and C
    size_t adjacencyMatrix_mem_size = sizeof(int) * NUM_VERTICES * NUM_VERTICES;
    int *adjacencyMatrix = (int *) malloc(adjacencyMatrix_mem_size);
    int *independentSet = (int *) malloc(NUM_VERTICES * sizeof(int));

    if (adjacencyMatrix == NULL || independentSet == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrices!\n");
        exit(EXIT_FAILURE);
    }

    //randomGraph(adjacencyMatrix);
    randomGraph(adjacencyMatrix);

    if (SEQUENTIAL) {
        lfIndependentSet(adjacencyMatrix, independentSet);
    } else {
        printf("Computing result using CUDA Kernel...\n");
        
        float randoms[PSEUDO_RANDOM_SIZE];
        fillPseudoRandoms(randoms);

        // Allocate device memory
        int *d_adjacencyMatrix, *d_degrees, *d_independentSet, *d_marked, *d_removedNodes,
            *d_existsNonRemoved;

        float *d_randoms;

        /** Feel free to use other structures! **/
        cudaCheck(hipMalloc((void **) &d_adjacencyMatrix, adjacencyMatrix_mem_size));
        cudaCheck(hipMalloc((void **) &d_degrees, sizeof(int) * NUM_VERTICES));
        cudaCheck(hipMalloc((void **) &d_independentSet, sizeof(int) * NUM_VERTICES));
        cudaCheck(hipMalloc((void **) &d_marked, sizeof(int) * NUM_VERTICES));
        cudaCheck(hipMalloc((void **) &d_removedNodes, sizeof(int) * NUM_VERTICES));
        cudaCheck(hipMalloc((void **) &d_existsNonRemoved, sizeof(int)));

        cudaCheck(hipMalloc((void **) &d_randoms, sizeof(float) * PSEUDO_RANDOM_SIZE));

        // copy host memory to device
        cudaCheck(hipMemcpy(d_adjacencyMatrix, adjacencyMatrix, 
            adjacencyMatrix_mem_size, hipMemcpyHostToDevice));

        cudaCheck(hipMemcpy(d_randoms, &randoms, 
            sizeof(float) * PSEUDO_RANDOM_SIZE, hipMemcpyHostToDevice));

        /*
           Run your kernel(s) here
        */
        
        // Copy result from device to host
        cudaCheck(hipMemcpy(independentSet, d_independentSet, NUM_VERTICES * sizeof(int), 
            hipMemcpyDeviceToHost));

        hipFree(d_adjacencyMatrix);
        hipFree(d_degrees);
        hipFree(d_marked);
        hipFree(d_independentSet);
        hipFree(d_removedNodes);
    }


    printf("Checking computed result for correctness: ");
    
    if (PRINT_EDGES) printEdges(adjacencyMatrix);
    printArray(independentSet);

    bool correct = verifyMaximalIndependentSet(adjacencyMatrix, independentSet);

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(adjacencyMatrix);
    free(independentSet);

    if (correct) {
        return EXIT_SUCCESS;
    } else {
        return EXIT_FAILURE;
    }
}
